#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
    int device;
    hipDeviceProp_t props;
    
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);

    printf("Compute capability of device %d: %d.%d\n", device, props.major, props.minor);

    return 0;
}
